#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
void cutlass_semi_structured_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);
#endif

int32_t get_sm_version_num() {
  int32_t major_capability, minor_capability;
  hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor,
                         0);
  hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor,
                         0);
  int32_t version_num = major_capability * 10 + minor_capability;
  return version_num;
}

void cutlass_semi_structured_mm(torch::Tensor& c, torch::Tensor const& a,
                       torch::Tensor const& b) {
  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) == b.size(0) &&
              b.size(1) == c.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1 && c.stride(1) == 1);  // Row-major
  TORCH_CHECK(b.stride(0) == 1);                      // Column-major
  TORCH_CHECK(c.stride(0) % 16 == 0 &&
              b.stride(1) % 16 == 0);  // 16 Byte Alignment

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));
  int32_t version_num = get_sm_version_num();
  // Hopper

  // TODO: Guard against compilation issues for sm90 kernels
// #if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
  if (version_num >= 90) {
    cutlass_semi_structured_mm_sm90(c, a, b, a_scales, b_scales, bias);
    return;
  }
// #endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_semi_structured_mm for a compute capability less than "
      "CUDA device capability: ",
      version_num);
}
