#include "hip/hip_runtime.h"
// clang-format will break include orders
// clang-format off
#include <cudaTypedefs.h>

#if defined CUDA_VERSION && CUDA_VERSION >= 12000

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>

#include <iostream>
#include <sstream>
#include <vector>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"

#include "util/broadcast_load_epilogue_c3x.hpp"
#include "util/common.hpp"
// clang-format on

#include "util/common_gemm.cuh"

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_gemm_sm90_fp8_dispatch(torch::Tensor& out, torch::Tensor const& a,
                                    torch::Tensor const& e,
                                    torch::Tensor const& b,
                                    EpilogueArgs&&... args) {
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(e.dtype() == torch::kUInt8);
  TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

  using Cutlass3xGemmDefault =
      typename sm90_fp8_config_default<InType, OutType,
                                       Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM64 =
      typename sm90_fp8_config_M64<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM128 =
      typename sm90_fp8_config_M128<InType, OutType, Epilogue>::Cutlass3xGemm;

  uint32_t const m = a.size(0);
  uint32_t const mp2 =
      std::max(static_cast<uint32_t>(64), next_pow_2(m));  // next power of 2

  if (mp2 <= 64) {
    // m in [1, 64]
    return cutlass_test_gemm_caller<Cutlass3xGemmM64>(
        out, a, e, b, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 128) {
    // m in (64, 128]
    return cutlass_test_gemm_caller<Cutlass3xGemmM128>(
        out, a, e, b, std::forward<EpilogueArgs>(args)...);
  } else {
    // m in (128, inf)
    return cutlass_test_gemm_caller<Cutlass3xGemmDefault>(
        out, a, e, b, std::forward<EpilogueArgs>(args)...);
  }
}

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_gemm_sm90_int8_dispatch(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& e,
                                     torch::Tensor const& b,
                                     EpilogueArgs&&... args) {
  static_assert(std::is_same<InType, int8_t>());
  TORCH_CHECK(a.dtype() == torch::kInt8);
  TORCH_CHECK(e.dtype() == torch::kUInt8);
  TORCH_CHECK(b.dtype() == torch::kInt8);

  using Cutlass3xGemmDefault =
      typename sm90_int8_config_default<InType, OutType,
                                        Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM128 =
      typename sm90_int8_config_M128<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM64 =
      typename sm90_int8_config_M64<InType, OutType, Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM32NBig =
      typename sm90_int8_config_M32_NBig<InType, OutType,
                                         Epilogue>::Cutlass3xGemm;
  using Cutlass3xGemmM32NSmall =
      typename sm90_int8_config_M32_NSmall<InType, OutType,
                                           Epilogue>::Cutlass3xGemm;

  uint32_t const n = out.size(1);
  bool const is_small_n = n < 8192;

  uint32_t const m = a.size(0);
  uint32_t const mp2 =
      std::max(static_cast<uint32_t>(32), next_pow_2(m));  // next power of 2

  if (mp2 <= 32) {
    // m in [1, 32]
    if (is_small_n) {
      return cutlass_test_gemm_caller<Cutlass3xGemmM32NSmall>(
          out, a, e, b, std::forward<EpilogueArgs>(args)...);
    } else {
      return cutlass_test_gemm_caller<Cutlass3xGemmM32NBig>(
          out, a, e, b, std::forward<EpilogueArgs>(args)...);
    }
  } else if (mp2 <= 64) {
    // m in (32, 64]
    return cutlass_test_gemm_caller<Cutlass3xGemmM64>(
        out, a, e, b, std::forward<EpilogueArgs>(args)...);
  } else if (mp2 <= 128) {
    // m in (64, 128]
    return cutlass_test_gemm_caller<Cutlass3xGemmM128>(
        out, a, e, b, std::forward<EpilogueArgs>(args)...);
  } else {
    // m in (128, inf)
    return cutlass_test_gemm_caller<Cutlass3xGemmDefault>(
        out, a, e, b, std::forward<EpilogueArgs>(args)...);
  }
}

template <template <typename, typename, typename> typename Epilogue,
          typename... EpilogueArgs>
void cutlass_scaled_test_mm_sm90_epilogue(torch::Tensor& out, torch::Tensor const& a,
                                     torch::Tensor const& e,
                                     torch::Tensor const& b,
                                     EpilogueArgs&&... epilogue_args) {
  if (a.dtype() == torch::kInt8) {
    TORCH_CHECK(e.dtype() == torch::kUInt8);
    TORCH_CHECK(b.dtype() == torch::kInt8);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_sm90_int8_dispatch<int8_t, cutlass::bfloat16_t,
                                             Epilogue>(
          out, a, e, b, std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_sm90_int8_dispatch<int8_t, cutlass::half_t, Epilogue>(
          out, a, e, b, std::forward<EpilogueArgs>(epilogue_args)...);
    }
  } else {
    TORCH_CHECK(a.dtype() == torch::kFloat8_e4m3fn);
    TORCH_CHECK(e.dtype() == torch::kUInt8);
    TORCH_CHECK(b.dtype() == torch::kFloat8_e4m3fn);

    if (out.dtype() == torch::kBFloat16) {
      return cutlass_gemm_sm90_fp8_dispatch<cutlass::float_e4m3_t,
                                            cutlass::bfloat16_t, Epilogue>(
          out, a, e, b, std::forward<EpilogueArgs>(epilogue_args)...);
    } else {
      TORCH_CHECK(out.dtype() == torch::kFloat16);
      return cutlass_gemm_sm90_fp8_dispatch<cutlass::float_e4m3_t,
                                            cutlass::half_t, Epilogue>(
          out, a, e, b, std::forward<EpilogueArgs>(epilogue_args)...);
    }
  }
}

void cutlass_scaled_test_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& e,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);
  if (bias) {
    TORCH_CHECK(bias->dtype() == c.dtype(),
                "currently bias dtype must match output dtype ", c.dtype());
    return cutlass_scaled_test_mm_sm90_epilogue<ScaledEpilogueBias>(
        c, a, e, b, a_scales, b_scales, *bias);
  } else {
    return cutlass_scaled_test_mm_sm90_epilogue<ScaledEpilogue>(c, a, e, b,
                                                           a_scales,
                                                           b_scales);
  }
}

void cutlass_scaled_test_mm_azp_sm90(torch::Tensor& out, torch::Tensor const& a,
                                torch::Tensor const& e,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias) {
  TORCH_CHECK(a_scales.dtype() == torch::kFloat32);
  TORCH_CHECK(b_scales.dtype() == torch::kFloat32);

  if (azp) {
    return cutlass_scaled_test_mm_sm90_epilogue<ScaledEpilogueBiasAzpToken>(
        out, a, e, b, a_scales, b_scales, azp_adj, *azp, bias);
  } else {
    return cutlass_scaled_test_mm_sm90_epilogue<ScaledEpilogueBiasAzp>(
        out, a, e, b, a_scales, b_scales, azp_adj, bias);
  }
}

#endif
