#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
void cutlass_scaled_sparse_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                                   torch::Tensor const& e,
                                   torch::Tensor const& b,
                                   torch::Tensor const& a_scales,
                                   torch::Tensor const& b_scales,
                                   c10::optional<torch::Tensor> const& bias);
#endif

int32_t test_get_sm_version_num() {
  int32_t major_capability, minor_capability;
  hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor,
                         0);
  hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor,
                         0);
  int32_t version_num = major_capability * 10 + minor_capability;
  return version_num;
}

void cutlass_scaled_sparse_mm(torch::Tensor& c, torch::Tensor const& a,
                              torch::Tensor const& e, torch::Tensor const& b,
                              torch::Tensor const& a_scales,
                              torch::Tensor const& b_scales,
                              c10::optional<torch::Tensor> const& bias) {
  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) * 2 == b.size(0) &&
              b.size(1) == c.size(1));
  TORCH_CHECK(a_scales.numel() == 1 || a_scales.numel() == a.size(0));
  TORCH_CHECK(b_scales.numel() == 1 || b_scales.numel() == b.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1);                      // Row-major
  TORCH_CHECK(b.stride(0) == 1 && c.stride(0) == 1);  // Column-major
  TORCH_CHECK(c.stride(1) % 16 == 0);                 // 16 Byte Alignment
  TORCH_CHECK(b.stride(1) % 16 == 0);                 // 16 Byte Alignment
  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous() &&
                bias->dim() == 1);
  }

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));
  int32_t version_num = test_get_sm_version_num();
  // Hopper

  // Guard against compilation issues for sm90 kernels
#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
  if (version_num >= 90) {
    cutlass_scaled_sparse_mm_sm90(c, a, e, b, a_scales, b_scales, bias);
    return;
  }
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_sparse_mm for a compute capability less than "
      "CUDA device capability: ",
      version_num);
}
