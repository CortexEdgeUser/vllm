#include "hip/hip_runtime.h"
#include "marlin_moe_kernel.cuh"

namespace marlin_moe {

#define __CALL_IF_MOE_8(W_TYPE, THREAD_M_BLOCKS, THREAD_N_BLOCKS,            \
                      THREAD_K_BLOCKS, HAS_ACT_ORDER, HAS_ZP, GROUP_BLOCKS,   \
                      NUM_THREADS)                                            \
  else if (q_type == W_TYPE && thread_m_blocks == THREAD_M_BLOCKS &&          \
           thread_n_blocks == THREAD_N_BLOCKS &&                              \
           thread_k_blocks == THREAD_K_BLOCKS &&                              \
           has_act_order == HAS_ACT_ORDER && has_zp == HAS_ZP &&              \
           group_blocks == GROUP_BLOCKS && num_threads == NUM_THREADS) {      \
    hipFuncSetAttribute(MarlinMoE<W_TYPE.id(), NUM_THREADS, THREAD_M_BLOCKS, \
                                   THREAD_N_BLOCKS, THREAD_K_BLOCKS, STAGES,  \
                                   HAS_ACT_ORDER, HAS_ZP, GROUP_BLOCKS>,      \
                         hipFuncAttributeMaxDynamicSharedMemorySize,         \
                         max_shared_mem);                                     \
    MarlinMoE<W_TYPE.id(), NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS,     \
              THREAD_K_BLOCKS, STAGES, HAS_ACT_ORDER, HAS_ZP, GROUP_BLOCKS>   \
        <<<blocks, NUM_THREADS, max_shared_mem, stream>>>(                    \
            A_ptr, B_ptr, C_ptr, sorted_ids_ptr, topk_weights_ptr, s_ptr,     \
            zp_ptr, g_idx_ptr, expert_offsets_ptr, num_groups, expert_idx,    \
            num_experts, topk, prob_m, prob_n, prob_k, tot_m, locks,          \
            replicate_input, apply_weights, m_block, max_par,                 \
            cfg_max_m_blocks);                                                \
  }


#define AWQ_CALL_IF_MOE_8(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)             \
  __CALL_IF_MOE_8(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, true, -1, NUM_THREADS) \
  __CALL_IF_MOE_8(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, true, 2, NUM_THREADS)  \
  __CALL_IF_MOE_8(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, true, 4, NUM_THREADS)  \
  __CALL_IF_MOE_8(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, true, 8, NUM_THREADS)  \
                                                                             \
  __CALL_IF_MOE_8(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, true, -1, NUM_THREADS) \
  __CALL_IF_MOE_8(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, true, 2, NUM_THREADS)  \
  __CALL_IF_MOE_8(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, true, 4, NUM_THREADS)  \
  __CALL_IF_MOE_8(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, true, 8, NUM_THREADS)  \
                                                                             \
  __CALL_IF_MOE_8(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, true, -1, NUM_THREADS) \
  __CALL_IF_MOE_8(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, true, 2, NUM_THREADS)  \
  __CALL_IF_MOE_8(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, true, 4, NUM_THREADS)  \
  __CALL_IF_MOE_8(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, true, 8, NUM_THREADS)  \
                                                                             \
  __CALL_IF_MOE_8(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, true, -1, NUM_THREADS) \
  __CALL_IF_MOE_8(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, true, 2, NUM_THREADS)  \
  __CALL_IF_MOE_8(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, true, 4, NUM_THREADS)  \
  __CALL_IF_MOE_8(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, true, 8, NUM_THREADS)

// We return bool so we can create these different kernel calls as a sequence
// of if-elseif's.
bool call_marlin_moe_kernel_ku8(
        vllm::ScalarType const& q_type, int thread_m_blocks,
        int thread_n_blocks, int thread_k_blocks, bool has_act_order,
        bool has_zp, int group_blocks, int num_threads, int blocks,
        int max_shared_mem, hipStream_t stream, const int4* A_ptr,
        const int4* B_ptr, int4* C_ptr, const int* sorted_ids_ptr,
        const float* topk_weights_ptr, const int4* s_ptr, const int4* zp_ptr,
        const int* g_idx_ptr, int* expert_offsets_ptr, int num_groups,
        int expert_idx, int num_experts, int topk, int prob_m, int prob_n,
        int prob_k, int tot_m, int* locks, bool replicate_input,
        bool apply_weights, int m_block, int max_par, int cfg_max_m_blocks) {
    if (false) {
    }
    AWQ_CALL_IF_MOE_8(vllm::kU8, 16, 4, 256)
    AWQ_CALL_IF_MOE_8(vllm::kU8, 8, 8, 256)
    AWQ_CALL_IF_MOE_8(vllm::kU8, 8, 4, 128)
    AWQ_CALL_IF_MOE_8(vllm::kU8, 4, 8, 128)
    else {
        return false;
    }
    return true;
}

}  // namespace marlin_moe
