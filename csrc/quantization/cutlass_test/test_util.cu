#include <cudaTypedefs.h>

#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>

#include <iostream>
#include <sstream>
#include <vector>

#include "cutlass/cutlass.h"


#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"
#include "cutlass/numeric_types.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/detail/dependent_false.hpp"

#include "broadcast_load_epilogue_c3x.hpp"
#include "common.hpp"

#include "cutlass/transform/device/transform_universal_adapter.hpp"
#include "cutlass/transform/kernel/sparse_gemm_compressor.hpp"

#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include <iostream>

#include "cutlass/cutlass.h"

#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/dispatch_policy.hpp"

#include "host_tensor.h"
#include "packed_stride.hpp"

#include "helper.h"

#include "common_gemm.cuh"

/// Make A structured sparse by replacing elements with 0 and compress it
template<typename ElementA_>
bool sparsify_and_compress(torch::Tensor& a_compressed, torch::Tensor& e, torch::Tensor const& a)
{
  // Checks for conformality
  TORCH_CHECK(a.dtype() == torch::kInt8 ||
              a.dtype() == torch::kFloat8_e4m3fn);
  TORCH_CHECK(a.dim() == 2)
  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1)

  int m = a.size(0);
  int k = a.size(1);

  using ProblemShape = Shape<int,int,int,int>;
  using ElementA = ElementA_;
  using LayoutTagA = cutlass::layout::RowMajor;

  // Layouts for reference (non-sparse) tensors
  using StrideA = cutlass::gemm::TagToStrideA_t<LayoutTagA>;
  using StrideE = StrideA;

  using KernelSchedule =
      cutlass::gemm::KernelTmaWarpSpecializedFP8FastAccum;
  using EpilogueSchedule = typename cutlass::epilogue::TmaWarpSpecialized;
  using TileShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_1, _2, _1>;
  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    float, float,
    float, LayoutTagA, 4,
    float, LayoutTagA, 4,
    EpilogueSchedule
  >::CollectiveOp;

  // static constexpr size_t CEStorageSize =
  //     sizeof(typename CollectiveEpilogue::SharedStorage);
  // using Stages = typename cutlass::gemm::collective::StageCountAutoCarveout<
  //     static_cast<int>(CEStorageSize)>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
    ElementA, LayoutTagA, 32, // Assuming 8 bits - TODO: Extend to other types
    ElementA, cutlass::layout::ColumnMajor, 16,
    float,
    TileShape, ClusterShape,
    typename cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
  >::CollectiveOp;
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    ProblemShape,
    CollectiveMainloop,
    CollectiveEpilogue
  >;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using ElementE = typename Gemm::GemmKernel::CollectiveMainloop::ElementE;
  using SparseConfig = typename Gemm::GemmKernel::CollectiveMainloop::SparseConfig;

  typename Gemm::GemmKernel::ProblemShape prob_shape{m, 1, k, 1};

  // Offline compressor kernel
  using CompressorUtility = cutlass::transform::kernel::StructuredSparseCompressorUtility<
                            ProblemShape,
                            ElementA,
                            LayoutTagA,
                            SparseConfig>;

  using CompressorKernel = cutlass::transform::kernel::StructuredSparseCompressor<
                            ProblemShape,
                            ElementA,
                            LayoutTagA,
                            SparseConfig,
                            cutlass::arch::Sm90>;

  using Compressor = cutlass::transform::device::TransformUniversalAdapter<CompressorKernel>;
  
  auto [M, N, K, L] = prob_shape;

  StrideA stride_A;
  StrideA stride_A_compressed;
  StrideE stride_E;

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));

  CompressorUtility compressor_utility(prob_shape, stride_A);

  int ME = compressor_utility.get_metadata_m_physical();
  int KE = compressor_utility.get_metadata_k_physical();
  int KC = compressor_utility.get_tensorA_k_physical();

  auto a_ptr = static_cast<typename Gemm::ElementA*>(a.data_ptr());

  // cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
  // cutlass::DeviceAllocation<typename Gemm::ElementA> block_A_compressed;
  // cutlass::DeviceAllocation<typename Gemm::CollectiveMainloop::ElementE> block_E;

  auto a_compressed_ptr = static_cast<typename Gemm::ElementA*>(a_compressed.data_ptr());
  auto e_ptr = static_cast<typename Gemm::CollectiveMainloop::ElementE*>(e.data_ptr());

  // block_A_compressed.reset(M * KC * L);
  // block_E.reset(ME * KE * L);

  stride_A_compressed = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, KC, L));
  stride_E = cutlass::make_cute_packed_stride(StrideE{}, cute::make_shape(ME, KE, L));

  // Random sparsification is performed on host
  std::vector<ElementA> block_A_host(m * k);
  cutlass::device_memory::copy_to_host(block_A_host.data(), a_ptr, m * k);
  compressor_utility.structure_sparse_zero_mask_fill(block_A_host.data(), 2024);
  cutlass::device_memory::copy_to_device(a_ptr, block_A_host.data(), m * k);

  cutlass::KernelHardwareInfo hw_info;
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);
  typename Compressor::Arguments arguments {
    prob_shape,
    { a_ptr,
      stride_A,
      a_compressed_ptr,
      e_ptr },
    {hw_info} };

  Compressor compressor_op;
  size_t workspace_size = Compressor::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  CUTLASS_CHECK(compressor_op.can_implement(arguments));
  CUTLASS_CHECK(compressor_op.initialize(arguments, workspace.get()));
  CUTLASS_CHECK(compressor_op.run());
  CUDA_CHECK(hipDeviceSynchronize());

  return true;
}

bool cutlass_sparsify_and_compress_entry(torch::Tensor& a_compressed, torch::Tensor& e, torch::Tensor const& a)
{
  // if (a.dtype() == torch::kBFloat16) {
  //   return sparsify_and_compress<cutlass::bfloat16_t>(a_compressed, e, a);
  // } else if (a.dtype() == torch::kFloat16) {
  //   return sparsify_and_compress<cutlass::half_t>(a_compressed, e, a);
  // } else
  if (a.dtype() == torch::kFloat8_e4m3fn) {
    return sparsify_and_compress<cutlass::float_e4m3_t>(a_compressed, e, a);
  }
  // else if (a.dtype() == torch::kInt8) {
  //   return sparsify_and_compress<int8_t>(a_compressed, e, a);
  // }
  return false;
}