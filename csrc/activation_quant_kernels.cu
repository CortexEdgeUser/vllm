#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>
#include "core/math.hpp"
#include "cuda_compat.h"
#include "dispatch_utils.h"

using FP8_TYPE = c10::Float8_e4m3fn;
C10_HOST_DEVICE constexpr auto FP8_E4M3_MAX =
    std::numeric_limits<FP8_TYPE>::max();
// using FP8_TYPE = c10::Float8_e4m3fnuz;
namespace vllm {

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

template <bool is_scale_inverted>
__device__ __forceinline__ FP8_TYPE scaled_fp8_conversion(float const val,
                                                          float const scale) {
  float x = 0.0f;
  if constexpr (is_scale_inverted) {
    x = val * scale;
  } else {
    x = val / scale;
  }
  float r = fmax(-FP8_E4M3_MAX, fmin(x, FP8_E4M3_MAX));
  return static_cast<c10::Float8_e4m3fn>(r);
}

// Activation and gating kernel template.
template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void act_and_mul_quant_kernel(
    FP8_TYPE* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., 2, d]
    const float* scale, const int d) {
  const int32_t token_idx = blockIdx.x;
  const int32_t blocks_per_token = gridDim.y;

  const int32_t elems_per_128bit_load = (128 / 8) / sizeof(scalar_t);

  const int32_t tgt_elems_per_block = div_ceil(d, blocks_per_token);
  const int32_t elems_per_block =
      next_multiple_of(elems_per_128bit_load, tgt_elems_per_block);
  const int32_t block_start = blockIdx.y * elems_per_block;
  int32_t block_end = block_start + elems_per_block;
  block_end = block_end > d ? d : block_end;

  const scalar_t* __restrict__ x_ptr = input + token_idx * 2 * d;
  const scalar_t* __restrict__ y_ptr = input + token_idx * 2 * d + d;
  FP8_TYPE* __restrict__ out_ptr = out + token_idx * d;

  // 128-bit vectorized code
  const int32_t vec_loop_end =
      prev_multiple_of(elems_per_128bit_load, block_end);
  const int32_t vec_end_idx = vec_loop_end / elems_per_128bit_load;
  const int32_t vec_start_idx = block_start / elems_per_128bit_load;

  const int4* __restrict__ x_128bit_ptr = reinterpret_cast<const int4*>(x_ptr);
  const int4* __restrict__ y_128bit_ptr = reinterpret_cast<const int4*>(y_ptr);
  int2* __restrict__ out_128bit_ptr = reinterpret_cast<int2*>(out_ptr);

  float inverted_scale = 1 / *scale;
#pragma unroll
  for (int32_t vec_idx = vec_start_idx + threadIdx.x; vec_idx < vec_end_idx;
       vec_idx += blockDim.x) {
    const int4 x_128bit = VLLM_LDG(&x_128bit_ptr[vec_idx]);
    const int4 y_128bit = VLLM_LDG(&y_128bit_ptr[vec_idx]);
    using scalar_128bit_vec_t = std::array<scalar_t, elems_per_128bit_load>;
    using scalar_64bit_vec_t = std::array<FP8_TYPE, elems_per_128bit_load>;

    scalar_64bit_vec_t out_vec;
    const auto x_vec = reinterpret_cast<scalar_128bit_vec_t const&>(x_128bit);
    const auto y_vec = reinterpret_cast<scalar_128bit_vec_t const&>(y_128bit);

#pragma unroll
    for (int i = 0; i < elems_per_128bit_load; i++) {
      out_vec[i] = scaled_fp8_conversion<true>(ACT_FN(x_vec[i]) * y_vec[i],
                                               inverted_scale);
    }

    out_128bit_ptr[vec_idx] = reinterpret_cast<const int2&>(out_vec);
  }

  // Scalar cleanup code
  if (block_end > vec_loop_end) {
    for (int64_t idx = vec_loop_end + threadIdx.x; idx < block_end;
         idx += blockDim.x) {
      const scalar_t x = VLLM_LDG(&x_ptr[idx]);
      const scalar_t y = VLLM_LDG(&y_ptr[idx]);
      // out_ptr[idx] = ACT_FN(x) * y;
      out_ptr[idx] = scaled_fp8_conversion<true>(ACT_FN(x) * y, inverted_scale);
    }
  }
}
}  // namespace vllm

// Launch activation, gating, and quantize kernel.
#define LAUNCH_ACTIVATION_GATE_KERNEL(KERNEL)                           \
  int d = input.size(-1) / 2;                                           \
  int64_t num_tokens = input.numel() / input.size(-1);                  \
  dim3 grid(num_tokens, num_tokens > 16 ? num_tokens > 32 ? 1 : 2 : 4); \
  dim3 block(std::min(d, 512));                                         \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));     \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();         \
  VLLM_DISPATCH_FLOATING_TYPES(                                         \
      input.scalar_type(), "act_and_mul_kernel", [&] {                  \
        vllm::act_and_mul_quant_kernel<scalar_t, KERNEL<scalar_t>>      \
            <<<grid, block, 0, stream>>>(out.data_ptr<FP8_TYPE>(),      \
                                         input.data_ptr<scalar_t>(),    \
                                         scale.data_ptr<float>(), d);   \
      });

void silu_and_mul_quant(torch::Tensor& out,  // [..., d]
                        torch::Tensor& input,
                        torch::Tensor& scale)  // [..., 2 * d]
{
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::silu_kernel);
}